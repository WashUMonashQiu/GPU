#include "hip/hip_runtime.h"
#include "cuPrintf.cu"
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <string>

// --------------------------------------------------------------------------
// Helper functions
// --------------------------------------------------------------------------

// testcase datastructure
struct params {
  float xi[3]; float xj[3];
  float vi[3]; float vj[3];
  float omegai[3]; float omegaj[3];
  float radi; float radj;
  float rmassi; float rmassj;
  float massi; float massj;
  float shear[3]; float torque[3]; float force[3];
  float expected_shear[3]; float expected_torque[3]; float expected_force[3];
};

struct params parse_csv_string(const char *str) {
  struct params result;
  sscanf(str,
    "%f, %f, %f, %f, %f, %f, "                //x
    "%f, %f, %f, %f, %f, %f, "                //v
    "%f, %f, %f, %f, %f, %f, "                //omega
    "%f, %f, "                                //radius
    "%f, %f, "                                //rmass
    "%f, %f, %f, %f, %f, %f, %f, %f, %f, "    //shear, torque, force
    "%f, %f, %f, %f, %f, %f, %f, %f, %f, ",   //*expected* shear, torque, force
    &result.xi[0], &result.xi[1], &result.xi[2],
    &result.xj[0], &result.xj[1], &result.xj[2],
    &result.vi[0], &result.vi[1], &result.vi[2],
    &result.vj[0], &result.vj[1], &result.vj[2],
    &result.omegai[0], &result.omegai[1], &result.omegai[2],
    &result.omegaj[0], &result.omegaj[1], &result.omegaj[2],
    &result.radi, &result.radj,
    &result.rmassi, &result.rmassj,
    &result.shear[0], &result.shear[1], &result.shear[2],
    &result.torque[0], &result.torque[1], &result.torque[2],
    &result.force[0], &result.force[1], &result.force[2],
    &result.expected_shear[0], &result.expected_shear[1], &result.expected_shear[2],
    &result.expected_torque[0], &result.expected_torque[1], &result.expected_torque[2],
    &result.expected_force[0], &result.expected_force[1], &result.expected_force[2]
    );
  result.massi = result.massj = 0;
  return result;
}

bool check_result_vector(const char* id, float expected[3], float actual[3], const float epsilon) {
  static bool verbose = false;
  bool flag = (fabs(expected[0] - actual[0]) > epsilon ||
               fabs(expected[1] - actual[1]) > epsilon ||
               fabs(expected[2] - actual[2]) > epsilon);
  const char *marker = flag ? "***" : "   ";

  if (flag || verbose) {
    printf("%s%s: {%f, %f, %f} / {%f, %f, %f}%s\n",
        marker,
        id,
        expected[0], expected[1], expected[2],
        actual[0], actual[1], actual[2],
        marker
        );
  }
  return flag;
}

// --------------------------------------------------------------------------
// GPU Kernel
// --------------------------------------------------------------------------
#define sqrtFiveOverSix 0.91287092917527685576161630466800355658790782499663875
__global__ void pair_interaction(
  //inputs
    float *xi, float *xj,           //position
    float *vi, float *vj,           //velocity
    float *omegai, float *omegaj,   //rotational velocity
    float radi, float radj,         //radius
    float rmassi, float rmassj,     //] mass
    float massi, float massj,       //]
    int typei, int typej,           //type
    float dt,                       //timestep

  //contact model parameters inputs
    int num_atom_types,
    float *Yeff,
    float *Geff,
    float *betaeff,
    float *coeffFrict,
    float nktv2p,

  //inouts
    float *shear,
    float *torque,
    float *force) {

  // del is the vector from j to i
  float delx = xi[0] - xj[0];
  float dely = xi[1] - xj[1];
  float delz = xi[2] - xj[2];

  float rsq = delx*delx + dely*dely + delz*delz;
  float radsum = radi + radj;
  if (rsq >= radsum*radsum) {
    //unset non-touching atoms
  } else {
    //distance between centres of atoms i and j
    //or, magnitude of del vector
    float r = sqrt(rsq);
    float rinv = 1.0/r;
    float rsqinv = 1.0/rsq;
	
    // relative translational velocity
    float vr1 = vi[0] - vj[0];
    float vr2 = vi[1] - vj[1];
    float vr3 = vi[2] - vj[2];

    // normal component
    float vnnr = vr1*delx + vr2*dely + vr3*delz;
    float vn1 = delx*vnnr * rsqinv;
    float vn2 = dely*vnnr * rsqinv;
    float vn3 = delz*vnnr * rsqinv;

    // tangential component
    float vt1 = vr1 - vn1;
    float vt2 = vr2 - vn2;
    float vt3 = vr3 - vn3;

    // relative rotational velocity
    float wr1 = (radi*omegai[0] + radj*omegaj[0]) * rinv;
    float wr2 = (radi*omegai[1] + radj*omegaj[1]) * rinv;
    float wr3 = (radi*omegai[2] + radj*omegaj[2]) * rinv;

    // normal forces = Hookian contact + normal velocity damping
    float mi,mj;
    if (rmassi && rmassj) {
      mi=rmassi;
      mj=rmassj;
    } else if (massi && massj) {
      mi=massi;
      mj=massj;
    } else {
      //this should never fire
      return;
    }
    float meff = mi*mj/(mi+mj);
    //not-implemented: freeze_group_bit

    float deltan = radsum-r;

    //derive contact model parameters (inlined)
    //Yeff, Geff, betaeff, coeffFrict are lookup tables
    //todo: put these in shared memory
    int typeij = typei + (typej * num_atom_types);
    float reff = radi * radj / (radi + radj);
    float sqrtval = sqrt(reff * deltan);
    float Sn = 2.    * Yeff[typeij] * sqrtval;
    float St = 8.    * Geff[typeij] * sqrtval;
    float kn = 4./3. * Yeff[typeij] * sqrtval;
    float kt = St;
    float gamman=-2.*sqrtFiveOverSix*betaeff[typeij]*sqrt(Sn*meff);
    float gammat=-2.*sqrtFiveOverSix*betaeff[typeij]*sqrt(St*meff);
    float xmu=coeffFrict[typeij];
    kn /= nktv2p;
    kt /= nktv2p;

    //if dampflag gammat = 0
    float damp = gamman*vnnr*rsqinv;
	  float ccel = kn*(radsum-r)*rinv - damp;

    //not-implemented cohesionflag

    // relative velocities
    float vtr1 = vt1 - (delz*wr2-dely*wr3);
    float vtr2 = vt2 - (delx*wr3-delz*wr1);
    float vtr3 = vt3 - (dely*wr1-delx*wr2);

    // shear history effects
    shear[0] += vtr1 * dt;
    shear[1] += vtr2 * dt;
    shear[2] += vtr3 * dt;

    // rotate shear displacements
    float rsht = shear[0]*delx + shear[1]*dely + shear[2]*delz;
    rsht *= rsqinv;

    shear[0] -= rsht*delx;
    shear[1] -= rsht*dely;
    shear[2] -= rsht*delz;

    // tangential forces = shear + tangential velocity damping
    float fs1 = - (kt*shear[0] + gammat*vtr1);
    float fs2 = - (kt*shear[1] + gammat*vtr2);
    float fs3 = - (kt*shear[2] + gammat*vtr3);

    // rescale frictional displacements and forces if needed
    float fs = sqrt(fs1*fs1 + fs2*fs2 + fs3*fs3);
    float fn = xmu * fabs(ccel*r);
    if (fs > fn) {
      float shrmag = sqrt(shear[0]*shear[0] +
                          shear[1]*shear[1] +
                          shear[2]*shear[2]);
      if (shrmag != 0.0) {
        shear[0] = (fn/fs) * (shear[0] + gammat*vtr1/kt) - gammat*vtr1/kt;
        shear[1] = (fn/fs) * (shear[1] + gammat*vtr2/kt) - gammat*vtr2/kt;
        shear[2] = (fn/fs) * (shear[2] + gammat*vtr3/kt) - gammat*vtr3/kt;
        fs1 *= fn/fs;
        fs2 *= fn/fs;
        fs3 *= fn/fs;
      } else {
        fs1 = fs2 = fs3 = 0.0;
      }
    }

    float tor1 = rinv * (dely*fs3 - delz*fs2);
    float tor2 = rinv * (delz*fs1 - delx*fs3);
    float tor3 = rinv * (delx*fs2 - dely*fs1);

    // this is what we've been working up to!
    force[0] += delx*ccel + fs1;
    force[1] += dely*ccel + fs2;
    force[2] += delz*ccel + fs3;

    torque[0] -= radi*tor1;
    torque[1] -= radi*tor2;
    torque[2] -= radi*tor3;
  }
}

// --------------------------------------------------------------------------
// Main
// --------------------------------------------------------------------------
int main(void) {
  // Stiffness lookup tables (indexed on atom type)
  float Yeff[2][2];
  float Geff[2][2];
  float betaeff[2][2];
  float coeffFrict[2][2];

  // Inputs fixed across testcases
  float dt = 0.000010;
  Yeff[1][1] = 3134796.238245;
  Geff[1][1] = 556173.526140;
  betaeff[1][1] = -0.357857;
  coeffFrict[1][1] = 0.500000;
  float nktv2p = 1.000000;
  int typei = 1;
  int typej = 1;
  int max_type = 2;

  // Device versions of inputs
  float *d_xi; float *d_xj;
  float *d_vi; float *d_vj;
  float *d_omegai; float *d_omegaj;
  // not required for float inputs (rad, rmass, mass, type and dt)
  float *d_Yeff;
  float *d_Geff;
  float *d_betaeff;
  float *d_coeffFrict;
  float *d_shear;
  float *d_torque;
  float *d_force;

  // Device allocation of input parameters
  hipMalloc((void**)&d_xi, sizeof(float)*3);
  hipMalloc((void**)&d_xj, sizeof(float)*3);
  hipMalloc((void**)&d_vi, sizeof(float)*3);
  hipMalloc((void**)&d_vj, sizeof(float)*3);
  hipMalloc((void**)&d_omegai, sizeof(float)*3);
  hipMalloc((void**)&d_omegaj, sizeof(float)*3);
  hipMalloc((void**)&d_Yeff, sizeof(float)*max_type*max_type);
  hipMalloc((void**)&d_Geff, sizeof(float)*max_type*max_type);
  hipMalloc((void**)&d_betaeff, sizeof(float)*max_type*max_type);
  hipMalloc((void**)&d_coeffFrict, sizeof(float)*max_type*max_type);
  hipMalloc((void**)&d_shear, sizeof(float)*3);
  hipMalloc((void**)&d_torque, sizeof(float)*3);
  hipMalloc((void**)&d_force, sizeof(float)*3);

  // Flatten 2D lookup tables into contiguous memory
  float *h_Yeff = (float *)malloc(sizeof(float)*max_type*max_type);
  float *h_Geff = (float *)malloc(sizeof(float)*max_type*max_type);
  float *h_betaeff = (float *)malloc(sizeof(float)*max_type*max_type);
  float *h_coeffFrict = (float *)malloc(sizeof(float)*max_type*max_type);
  for (int i=0; i<max_type; i++) {
    for (int j=0; j<max_type; j++) {
      h_Yeff[i + (j*max_type)] = Yeff[i][j];
      h_Geff[i + (j*max_type)] = Geff[i][j];
      h_betaeff[i + (j*max_type)] = betaeff[i][j];
      h_coeffFrict[i + (j*max_type)] = coeffFrict[i][j];
    }
  }
  hipMemcpy(d_Yeff, h_Yeff, sizeof(float)*max_type*max_type, hipMemcpyHostToDevice);
  hipMemcpy(d_Geff, h_Geff, sizeof(float)*max_type*max_type, hipMemcpyHostToDevice);
  hipMemcpy(d_betaeff, h_betaeff, sizeof(float)*max_type*max_type, hipMemcpyHostToDevice);
  hipMemcpy(d_coeffFrict, h_coeffFrict, sizeof(float)*max_type*max_type, hipMemcpyHostToDevice);

  // Open testcase datafile
  std::ifstream data("pairwise_data.csv", std::fstream::in);
  if (!data.is_open()) {
    printf("Could not find/open file [pairwise_data.csv]\n");
    exit(-1);
  }

  // Test loop over datafile
  std::string line;
  while(std::getline(data, line)) {
    struct params testcase = parse_csv_string(line.c_str());
    hipMemcpy(d_xi, testcase.xi, sizeof(float)*3, hipMemcpyHostToDevice);
    hipMemcpy(d_xj, testcase.xj, sizeof(float)*3, hipMemcpyHostToDevice);
    hipMemcpy(d_vi, testcase.vi, sizeof(float)*3, hipMemcpyHostToDevice);
    hipMemcpy(d_vj, testcase.vj, sizeof(float)*3, hipMemcpyHostToDevice);
    hipMemcpy(d_omegai, testcase.omegai, sizeof(float)*3, hipMemcpyHostToDevice);
    hipMemcpy(d_omegaj, testcase.omegaj, sizeof(float)*3, hipMemcpyHostToDevice);
    hipMemcpy(d_shear, testcase.shear, sizeof(float)*3, hipMemcpyHostToDevice);
    hipMemcpy(d_torque, testcase.torque, sizeof(float)*3, hipMemcpyHostToDevice);
    hipMemcpy(d_force, testcase.force, sizeof(float)*3, hipMemcpyHostToDevice);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      printf("pre-kernel err is %s.\n", hipGetErrorString(err));
      exit(-1);
    }

    pair_interaction<<<1,1>>>(
      d_xi,d_xj,
      d_vi,d_vj,
      d_omegai, d_omegaj,
      testcase.radi, testcase.radj,
      testcase.rmassi, testcase.rmassj,
      testcase.massi, testcase.massj,
      typei, typej,
      dt,

      max_type,
      d_Yeff,
      d_Geff,
      d_betaeff,
      d_coeffFrict,
      nktv2p,

      d_shear, d_torque, d_force);

    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
      printf("post-kernel err is %s.\n", hipGetErrorString(err));
      exit(-1);
    }

    // Check results
    float shear[3];
    float torque[3];
    float force[3];
    hipMemcpy(shear, d_shear, sizeof(float)*3, hipMemcpyDeviceToHost);
    hipMemcpy(torque, d_torque, sizeof(float)*3, hipMemcpyDeviceToHost);
    hipMemcpy(force, d_force, sizeof(float)*3, hipMemcpyDeviceToHost);

    const float epsilon = 0.00001;
    check_result_vector("shear ", testcase.expected_shear, shear, epsilon);
    check_result_vector("torque", testcase.expected_torque, torque, epsilon);
    check_result_vector("force ", testcase.expected_force, force, epsilon);
  }

  return 0;
}
